#include "hip/hip_runtime.h"
#include <stdio.h>
#include <include/labwork.h>
#include <hip/hip_runtime_api.h>
#include <omp.h>

#define ACTIVE_THREADS 4

int main(int argc, char **argv) {
    printf("USTH ICT Master 2018, Advanced Programming for HPC.\n");
    if (argc < 2) {
        printf("Usage: labwork <lwNum> <inputImage>\n");
        printf("   lwNum        labwork number\n");
        printf("   inputImage   the input file name, in JPEG format\n");
        return 0;
    }

    int lwNum = atoi(argv[1]);
    std::string inputFilename;

    // pre-initialize CUDA to avoid incorrect profiling
    printf("Warming up...\n");
    char *temp;
    hipMalloc(&temp, 1024);

    Labwork labwork;
    if (lwNum != 2 ) {
        inputFilename = std::string(argv[2]);
        labwork.loadInputImage(inputFilename);
    }

    printf("Starting labwork %d\n", lwNum);
    Timer timer;
    timer.start();
    switch (lwNum) {
        case 1:
            labwork.labwork1_CPU();
            labwork.saveOutputImage("labwork2-cpu-out.jpg");
            printf("labwork 1 CPU ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            timer.start();
            labwork.labwork1_OpenMP();
            labwork.saveOutputImage("labwork2-openmp-out.jpg");
            break;
        case 2:
            labwork.labwork2_GPU();
            break;
        case 3:
            labwork.labwork3_GPU();
            labwork.saveOutputImage("labwork3-gpu-out.jpg");
            break;
        case 4:
            labwork.labwork4_GPU();
            labwork.saveOutputImage("labwork4-gpu-out.jpg");
            break;
        case 5:
            labwork.labwork5_CPU();
	printf("labwork 5 CPU ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork5-cpu-out.jpg");
	timer.start();
            labwork.labwork5_GPU();
	printf("labwork 5 GPU nonshared ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
	    labwork.saveOutputImage("labwork5-gpu-out.jpg");
	timer.start();
            labwork.labwork5_GPU2();
	printf("labwork 5 GPU shared ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork5-gpu2-out.jpg");
            break;
        case 6:
            labwork.labwork6_GPU();
            labwork.saveOutputImage("labwork6-gpu-out.jpg");
            break;
        case 7:
            labwork.labwork7_GPU();
            labwork.saveOutputImage("labwork7-gpu-out.jpg");
            break;
        case 8:
            labwork.labwork8_GPU();
            labwork.saveOutputImage("labwork8-gpu-out.jpg");
            break;
        case 9:
            labwork.labwork9_GPU();
            labwork.saveOutputImage("labwork9-gpu-out.jpg");
            break;
        case 10:
            labwork.labwork10_GPU();
            labwork.saveOutputImage("labwork10-gpu-out.jpg");
            break;
    }
    printf("labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
}

void Labwork::loadInputImage(std::string inputFileName) {
    inputImage = jpegLoader.load(inputFileName);
}

void Labwork::saveOutputImage(std::string outputFileName) {
    jpegLoader.save(outputFileName, outputImage, inputImage->width, inputImage->height, 90);
}

void Labwork::labwork1_CPU() {
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    for (int j = 0; j < 100; j++) {		// let's do it 100 times, otherwise it's too fast!
        for (int i = 0; i < pixelCount; i++) {
            outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] + (int) inputImage->buffer[i * 3 + 1] +
                                          (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
        }
    }
}

void Labwork::labwork1_OpenMP() {
	int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
#pragma omp parallel for schedule (dynamic)
    for (int j = 0; j < 100; j++) {		// let's do it 100 times, otherwise it's too fast!
        for (int i = 0; i < pixelCount; i++) {
            outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] + (int) inputImage->buffer[i * 3 + 1] +
                                          (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
        }
    }
}

int getSPcores(hipDeviceProp_t devProp) {
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch (devProp.major) {
        case 2: // Fermi
            if (devProp.minor == 1) cores = mp * 48;
            else cores = mp * 32;
            break;
        case 3: // Kepler
            cores = mp * 192;
            break;
        case 5: // Maxwell
            cores = mp * 128;
            break;
        case 6: // Pascal
            if (devProp.minor == 1) cores = mp * 128;
            else if (devProp.minor == 0) cores = mp * 64;
            else printf("Unknown device type\n");
            break;
        default:
            printf("Unknown device type\n");
            break;
    }
    return cores;
}

void Labwork::labwork2_GPU() {
    	int numberGPUs = 0;
	hipGetDeviceCount(&numberGPUs);
	printf("Number of GPUs: %d \n", numberGPUs);
	for (int i = 0; i< numberGPUs; i++){
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("GPU #%d \n" , i);
		printf(" - GPU name: %s \n", prop.name);
		printf(" - Clock rate: %d \n", prop.clockRate);
		printf(" - Number of cores: %d \n", getSPcores(prop));
		printf(" - Number of multiprocessors: %d \n", prop.multiProcessorCount);
		printf(" - Warp size: %d \n", prop.warpSize);
		printf(" - Memory clock rate: %d \n", prop.memoryClockRate);
		printf(" - Memory bus width: %d \n", prop.memoryBusWidth);
	}

}

__global__ void grayscale(uchar3 *input, uchar3 *output) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned char g = (input[tid].x + input[tid].y + input[tid].z) / 3;
	output[tid].z = output[tid].y = output[tid].x = g; //store in the register -> faster
}

void Labwork::labwork3_GPU() {
	
	int pixelCount = inputImage->width * inputImage->height;	
	int blockSize = 1024;
	int numBlock = pixelCount / blockSize;
	uchar3 *devInput,*devOutput;
	outputImage = static_cast<char *>(malloc(pixelCount * 3));
	hipMalloc(&devInput, pixelCount * 3);
	hipMalloc(&devOutput, pixelCount * 3);
	hipMemcpy(devInput, inputImage->buffer, pixelCount*3, hipMemcpyHostToDevice);
	grayscale<<<numBlock, blockSize>>>(devInput, devOutput);
	hipMemcpy(outputImage, devOutput, pixelCount*3, hipMemcpyDeviceToHost);
	hipFree(devInput);
	hipFree(devOutput);

}

__global__ void grayscale2d(uchar3* input, uchar3* output, int width, int height) {
    int tidx = threadIdx.x + blockIdx.x * blockDim.x;
	if (tidx >= width) return;
    int tidy = threadIdx.y + blockIdx.y * blockDim.y;
	if (tidy >= height) return;
    int tid = tidx + tidy * width; //gridDim.x*blockDim.x != width
    output[tid].x = (input[tid].x + input[tid].y + input[tid].z) / 3;
    output[tid].z = output[tid].y = output[tid].x; //load global mem too much -> slow
}
void Labwork::labwork4_GPU() {
	int pixelCount = inputImage->width * inputImage->height;	
	//int blockSize = 1024;
	dim3 blockSize = dim3(1024,1);
	//int numBlock = pixelCount / blockSize;
	//dim3 gridSize = dim3((pixelCount / (blockSize.x*blockSize.y))/2, 2);
	dim3 gridSize = dim3((inputImage->width + blockSize.x -1) / blockSize.x, (inputImage->height + blockSize.y -1) / blockSize.y);
	uchar3 *devInput,*devOutput;
	outputImage = static_cast<char *>(malloc(pixelCount * 3));
	hipMalloc(&devInput, pixelCount * 3);
	hipMalloc(&devOutput, pixelCount * 3);
	hipMemcpy(devInput, inputImage->buffer, pixelCount*3, hipMemcpyHostToDevice);
	grayscale2d<<<gridSize, blockSize>>>(devInput, devOutput,inputImage->width, inputImage->height);
	hipMemcpy(outputImage, devOutput, pixelCount*3, hipMemcpyDeviceToHost);
	hipFree(devInput);
	hipFree(devOutput);
}

// CPU implementation of Gaussian Blur
void Labwork::labwork5_CPU() {
    int kernel[] = { 0, 0, 1, 2, 1, 0, 0,  
                     0, 3, 13, 22, 13, 3, 0,  
                     1, 13, 59, 97, 59, 13, 1,  
                     2, 22, 97, 159, 97, 22, 2,  
                     1, 13, 59, 97, 59, 13, 1,  
                     0, 3, 13, 22, 13, 3, 0,
                     0, 0, 1, 2, 1, 0, 0 };
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = (char*) malloc(pixelCount * sizeof(char) * 3);
    for (int row = 0; row < inputImage->height; row++) {
        for (int col = 0; col < inputImage->width; col++) {
            int sum = 0;
            int c = 0;
            for (int y = -3; y <= 3; y++) {
                for (int x = -3; x <= 3; x++) {
                    int i = col + x;
                    int j = row + y;
                    if (i < 0) continue;
                    if (i >= inputImage->width) continue;
                    if (j < 0) continue;
                    if (j >= inputImage->height) continue;
                    int tid = j * inputImage->width + i;
                    unsigned char gray = (inputImage->buffer[tid * 3] + inputImage->buffer[tid * 3 + 1] + inputImage->buffer[tid * 3 + 2])/3;
                    int coefficient = kernel[(y+3) * 7 + x + 3];
                    sum = sum + gray * coefficient;
                    c += coefficient;
                }
            }
            sum /= c;
            int posOut = row * inputImage->width + col;
            outputImage[posOut * 3] = outputImage[posOut * 3 + 1] = outputImage[posOut * 3 + 2] = sum;
        }
    }
}

__global__ void nonsharedblur(uchar3* input, uchar3* output, int width, int height) {
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;
	if (tidx >= width) return;
	int tidy = threadIdx.y + blockIdx.y * blockDim.y;
	if (tidy >= height) return;
	int tid = tidx + tidy * width; //gridDim.x*blockDim.x != width
	/*	
	unsigned char g = input[tid].x;
	int sum = 0;
	avg blur
	sum +=input[tidx + tidy * width].x +
		input[(tidx -1) + (tidy -1) * width].x
		input[(tidx ) + (tidy -1) * width].x
		input[(tidx +1) + (tidy -1) * width].x
		input[(tidx -1) + (tidy ) * width].x
		input[(tidx +1) + (tidy ) * width].x
		input[(tidx -1) + (tidy +1) * width].x
		input[(tidx ) + (tidy +1) * width].x
		input[(tidx +1) + (tidy +1) * width].x
	sum /= 9;
	output[tidx + tidy*width].x = output[tidx + tidy*width].y = output[tidx + tidy*width].z = sum;
	*/
	int kernel[] = { 0, 0, 1, 2, 1, 0, 0,  
	 0, 3, 13, 22, 13, 3, 0,  
	 1, 13, 59, 97, 59, 13, 1,  
	 2, 22, 97, 159, 97, 22, 2,  
	 1, 13, 59, 97, 59, 13, 1,  
	 0, 3, 13, 22, 13, 3, 0,
	 0, 0, 1, 2, 1, 0, 0 };
	
	int sum = 0;
    	int c = 0;
	for (int row = -3; row<3; row++){
		for (int col = -3; col <3; col++){
		    int i = tidx + col;
		    int j = tidy + row;
		    if (i < 0) continue;
		    if (i >= width) continue;
		    if (j < 0) continue;
		    if (j >= height) continue;
		    int tid = j * width + i;
		    unsigned char g = (input[tid].x + input[tid].y + input[tid].z)/3;
		    int coefficient = kernel[(row+3) * 7 + col + 3];
		    sum = sum + g * coefficient;
		    c += coefficient;
		}
	}
	sum /= c;
    	output[tid].z = output[tid].y = output[tid].x = sum;
	//int posOut = tidy * width + tidx;
        //output[posOut * 3] = output[posOut * 3 + 1] = output[posOut * 3 + 2] = sum;
        
	
}

void Labwork::labwork5_GPU() {
    int pixelCount = inputImage->width * inputImage->height;	
	dim3 blockSize = dim3(32,32);
	dim3 gridSize = dim3((inputImage->width + blockSize.x -1) / blockSize.x, (inputImage->height + blockSize.y -1) / blockSize.y);
	uchar3 *devInput,*devOutput;
	outputImage = static_cast<char *>(malloc(pixelCount * sizeof(uchar3)));
	hipMalloc(&devInput, pixelCount * sizeof(uchar3));
	hipMalloc(&devOutput, pixelCount * sizeof(uchar3));
	hipMemcpy(devInput, inputImage->buffer, pixelCount*sizeof(uchar3), hipMemcpyHostToDevice);
	nonsharedblur<<<gridSize, blockSize>>>(devInput, devOutput,inputImage->width, inputImage->height);
	hipMemcpy(outputImage, devOutput, pixelCount*sizeof(uchar3), hipMemcpyDeviceToHost);
	hipFree(devInput);
	hipFree(devOutput);
}


__global__ void sharedblur(uchar3* input, uchar3* output, int* kernel, int width, int height) {
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;
	if (tidx >= width) return;
	int tidy = threadIdx.y + blockIdx.y * blockDim.y;
	if (tidy >= height) return;
	int tid = tidx + tidy * width; 
	/*
	int kernel[] = { 0, 0, 1, 2, 1, 0, 0,  
	 0, 3, 13, 22, 13, 3, 0,  
	 1, 13, 59, 97, 59, 13, 1,  
	 2, 22, 97, 159, 97, 22, 2,  
	 1, 13, 59, 97, 59, 13, 1,  
	 0, 3, 13, 22, 13, 3, 0,
	 0, 0, 1, 2, 1, 0, 0 };
	*/
	__shared__ int sharedkernel[49];
	for (int i=0; i<49; i++){
		sharedkernel[i] = kernel[i];
	}
	__syncthreads();
	int sum = 0;
    	int c = 0;
	for (int row = -3; row<3; row++){
		for (int col = -3; col <3; col++){
			int i = tidx + col;
		    int j = tidy + row;
		    if (i < 0) continue;
		    if (i >= width) continue;
		    if (j < 0) continue;
		    if (j >= height) continue;
		    int tid = j * width + i;
		    unsigned char g = (input[tid].x + input[tid].y + input[tid].z)/3;
		    int coefficient = sharedkernel[(row+3) * 7 + col + 3];
		    sum = sum + g * coefficient;
		    c += coefficient;
		}
	}
	sum /= c;
    	output[tid].z = output[tid].y = output[tid].x = sum;
}

void Labwork::labwork5_GPU2() {
	int kernel[] = { 0, 0, 1, 2, 1, 0, 0,  
	 0, 3, 13, 22, 13, 3, 0,  
	 1, 13, 59, 97, 59, 13, 1,  
	 2, 22, 97, 159, 97, 22, 2,  
	 1, 13, 59, 97, 59, 13, 1,  
	 0, 3, 13, 22, 13, 3, 0,
	 0, 0, 1, 2, 1, 0, 0 };
	int *share;
    int pixelCount = inputImage->width * inputImage->height;	
	dim3 blockSize = dim3(32,32);
	dim3 gridSize = dim3((inputImage->width + blockSize.x -1) / blockSize.x, (inputImage->height + blockSize.y -1) / blockSize.y);
	uchar3 *devInput,*devOutput;
	outputImage = static_cast<char *>(malloc(pixelCount * sizeof(uchar3)));
	hipMalloc(&devInput, pixelCount * sizeof(uchar3));
	hipMalloc(&devOutput, pixelCount * sizeof(uchar3));
	hipMalloc(&share, sizeof(kernel));
	hipMemcpy(devInput, inputImage->buffer, pixelCount*sizeof(uchar3), hipMemcpyHostToDevice);
	hipMemcpy(share, kernel, sizeof(kernel), hipMemcpyHostToDevice);
	sharedblur<<<gridSize, blockSize>>>(devInput, devOutput, share, inputImage->width, inputImage->height);
	hipMemcpy(outputImage, devOutput, pixelCount*sizeof(uchar3), hipMemcpyDeviceToHost);
	hipFree(devInput);
	hipFree(devOutput);
	hipFree(share);
}


void Labwork::labwork6_GPU() {

}

void Labwork::labwork7_GPU() {

}

void Labwork::labwork8_GPU() {

}

void Labwork::labwork9_GPU() {

}

void Labwork::labwork10_GPU() {

}
